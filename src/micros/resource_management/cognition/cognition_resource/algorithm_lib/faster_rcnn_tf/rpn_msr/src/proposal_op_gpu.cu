
#include <hip/hip_runtime.h>
#if GOOGLE_CUDA

#define EIGEN_USE_GPU

#include <iostream>
#include <cfloat>
#include <cuda.h>

#include "third_party/eigen3/unsupported/Eigen/CXX11/Tensor"
#include "tensorflow/core/framework/op_kernel.h"

#include "fast_rcnn/config.h"
#include "fast_rcnn/bbox_transform.h"
#include <nms/gpu_nms.h>

using namespace std;
using namespace tensorflow;

typedef Eigen::Tensor<float, 1, Eigen::RowMajor> Tensor1f;
typedef Eigen::Tensor<float, 2, Eigen::RowMajor> Tensor2f;
typedef Eigen::Tensor<float, 3, Eigen::RowMajor> Tensor3f;
typedef Eigen::Tensor<float, 4, Eigen::RowMajor> Tensor4f;
typedef Eigen::Tensor<int, 1, Eigen::RowMajor> Tensor1i;

class anchors_gen
{
public:
    anchors_gen();
    ~anchors_gen();
    vector<vector<float> > ratio_enum(vector<float>);
    vector<float> whctrs(vector<float>);
    vector<float> mkanchor(float w,float h,float x_ctr,float y_ctr);
    vector<vector<float> > scale_enum(vector<float>);
    vector<vector<float> > generate_anchors();
private:
    int base_size;
    float ratios[3];
    float scales[3];
};

__global__ void ProposalKernel(const float *boxes_data, const float *deltas_data, int A, int k_height, int k_width, float height, float width, float stride, float min_pro, float *pred_boxes, char *masks)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if(i >= k_height * k_width * A) return;
    float box_x1 = boxes_data[i%A*4] + ((i/A)%(k_width)) * stride;
    float box_y1 = boxes_data[i%A*4+1] + ((i/A)/(k_width)) * stride;
    float box_x2 = boxes_data[i%A*4+2] + ((i/A)%(k_width)) * stride;
    float box_y2 = boxes_data[i%A*4+3] + ((i/A)/(k_width)) * stride;
    float width_box  = box_x2 - box_x1 + 1.0f;
    float height_box = box_y2 - box_y1 + 1.0f;
    float ctr_x = box_x1 + width_box * 0.5f;
    float ctr_y = box_y1 + height_box * 0.5f;
    float pred_w = std::exp(deltas_data[i*4+2]) * width_box;
    float pred_h = std::exp(deltas_data[i*4+3]) * height_box;
    float pred_ctr_x = deltas_data[i*4] * width_box + ctr_x;
    float pred_ctr_y = deltas_data[i*4+1] * height_box + ctr_y;
    pred_boxes[i*4] = pred_ctr_x - pred_w * 0.5f;
    if(pred_boxes[i*4] > width - 1)
    {
        pred_boxes[i*4] = width - 1;
    }
    else if(pred_boxes[i*4] < 0)
    {
        pred_boxes[i*4] = 0;
    }
    pred_boxes[i*4+1] = pred_ctr_y - pred_h * 0.5f;
    if(pred_boxes[i*4+1] > height - 1)
    {
        pred_boxes[i*4+1] = height - 1;
    }
    else if(pred_boxes[i*4+1] < 0)
    {
        pred_boxes[i*4+1] = 0;
    }
    pred_boxes[i*4+2] = pred_ctr_x + pred_w * 0.5f;
    if(pred_boxes[i*4+2] > width - 1)
    {
        pred_boxes[i*4+2] = width - 1;
    }
    else if(pred_boxes[i*4+2] < 0)
    {
        pred_boxes[i*4+2] = 0;
    }
    pred_boxes[i*4+3] = pred_ctr_y + pred_h * 0.5f;
    if(pred_boxes[i*4+3] > height - 1)
    {
        pred_boxes[i*4+3] = height - 1;
    }
    else if(pred_boxes[i*4+3] < 0)
    {
        pred_boxes[i*4+3] = 0;
    }
    float ws = pred_boxes[i*4+2] - pred_boxes[i*4] + 1.0f;
    float hs = pred_boxes[i*4+3] - pred_boxes[i*4+1] + 1.0f;
    if(ws >= min_pro && hs >= min_pro)
    {
        masks[i] = 1;
    }
    else
    {
        masks[i] = 0;
    }
}

void ProposalKernelLauncher(OpKernelContext* context, int feat_stride_, int cfg_key, const Eigen::GpuDevice &d)
{
    // Grab the input tensor
    const Tensor& rpn_cls_prob_reshape = context->input(0);//(1,h,w,18)
    //因为我们的预测值是anchors与gt的偏移量，这一层的值也就是detlas
    const Tensor& rpn_bbox_pred = context->input(1);//(1,h,w,36),
    const Tensor& im_info = context->input(2);//(1,3)
    // data should have 4 dimensions.
    OP_REQUIRES(context, rpn_cls_prob_reshape.dims() == 4,
                errors::InvalidArgument("data must be 4-dimensional"));
    // data should have 2 dimensions.
    OP_REQUIRES(context, rpn_bbox_pred.dims() == 4,
                errors::InvalidArgument("data must be 4-dimensional"));
    // data should have 2 dimensions.
    OP_REQUIRES(context, im_info.dims() == 2,
                errors::InvalidArgument("data must be 2-dimensional"));

    vector<vector<float> > text_anchor = anchors_gen().generate_anchors();
    const int num_anchors = text_anchor.size();

    const int height = rpn_cls_prob_reshape.dim_size(1);
    const int width  = rpn_cls_prob_reshape.dim_size(2);
    Tensor4f rpn_cls(1, height, width, 18);
    cudaMemcpy(rpn_cls.data(), rpn_cls_prob_reshape.flat<float>().data(), height * width * 18 * sizeof(float), cudaMemcpyHostToHost);
    Tensor4f bbox_deltas_t(1, height, width, 36);
    cudaMemcpy(bbox_deltas_t.data(), rpn_bbox_pred.flat<float>().data(), height * width * 36 * sizeof(float), cudaMemcpyHostToHost);
    Tensor2f im_shape(1,3);
    cudaMemcpy(im_shape.data(), im_info.flat<float>().data(), 3 * sizeof(float), cudaMemcpyHostToHost);

    int pre_nms_topN  = (cfg_key == 1 ? fast_rcnn::cfg.TRAIN.RPN_PRE_NMS_TOP_N : fast_rcnn::cfg.TEST.RPN_PRE_NMS_TOP_N);
    int post_nms_topN = (cfg_key == 1 ? fast_rcnn::cfg.TRAIN.RPN_POST_NMS_TOP_N : fast_rcnn::cfg.TEST.RPN_POST_NMS_TOP_N);
    float nms_thresh  = (cfg_key == 1 ? fast_rcnn::cfg.TRAIN.RPN_NMS_THRESH : fast_rcnn::cfg.TEST.RPN_NMS_THRESH);
    int min_size      = (cfg_key == 1 ? fast_rcnn::cfg.TRAIN.RPN_MIN_SIZE : fast_rcnn::cfg.TEST.RPN_MIN_SIZE);
    // the first set of _num_anchors channels are bg probs
    // the second set are the fg probs, which we want
    Eigen::array<int, 4> offsets({0, 0, 0, num_anchors});
    Eigen::array<int, 4> extents({1, height, width, num_anchors});
    Tensor4f scores_t = rpn_cls.slice(offsets, extents);
//*******************************************************************//
//和anchor_target_layer一样，也每个位置产生9个anchor，堆叠成anchors, (K×A, 4)，
//遍历顺序是先遍历完一个位置的所有anchor，然后宽度遍历，最后高度遍历，这种遍历顺序记作(h,w,a)
    const int A = num_anchors;
    const int K = height * width;

/********************************************************************/

    Eigen::array<int, 1> score_dims({K * A});
    Tensor1f scores_r = scores_t.reshape(score_dims);
    Tensor2f proposals_b(K * A, 4);
    std::vector<char> mask_vector(K * A);
    const int threads_per_block = d.maxCudaThreadsPerBlock();
    float *boxes_data = (float*)d.allocate(K * 4 * sizeof(float));
    float *deltas_data = (float*)d.allocate(K * A * 4 * sizeof(float));
    float *output = (float*)d.allocate(K * A * 4 * sizeof(float));
    char *masks = (char*)d.allocate(K * A);
    for(int i = 0; i < text_anchor.size(); i++)
    {
        d.memcpyHostToDevice(boxes_data + i * text_anchor[i].size(), text_anchor[i].data(), text_anchor[i].size() * sizeof(float));
    }
    d.memcpyHostToDevice(deltas_data, bbox_deltas_t.data(), K * A * 4 * sizeof(float));
    ProposalKernel<<<(K * A + threads_per_block - 1) / threads_per_block, threads_per_block, 0, d.stream()>>>(boxes_data, deltas_data, A, height, width, im_shape(0,0), im_shape(0,1), feat_stride_, min_size * im_shape(0,2), output, masks);
    d.memcpyDeviceToHost(proposals_b.data(), output, K * A * 4 * sizeof(float));
    d.memcpyDeviceToHost(mask_vector.data(), masks, K * A);
    d.synchronize();
    d.deallocate(boxes_data);
    d.deallocate(deltas_data);
    d.deallocate(output);
    d.deallocate(masks);
//Remove all boxes with any side smaller than min_size.
    std::vector<int> indices;
    for(int i = 0; i < K * A; i++)
    {
        if(mask_vector[i] == 1) indices.push_back(i);
    }
    int index_size = indices.size();
    Tensor2f proposals(index_size, 4);
    Tensor1f scores(index_size);
    for(int i = 0; i < index_size; i++)
    {
        proposals.chip(i,0) = proposals_b.chip(indices[i], 0);
        scores(i) = scores_r(indices[i]);
    }

/*****************************************************************/
//order是将scores展开，并由大到小排序的标号，先截取分数最高的pre_nms_topN个框，比如12000个（注意如果少于这个数就是全部），
//然后proposals和scores都按照这个顺序将框排好。这个时候的框已经没有(h,w,a)的遍历顺序了
    Tensor2f proposals_last_p(pre_nms_topN,4);
    Tensor1f scores_last_p(pre_nms_topN);

    vector<float> s;
    vector<vector<float>::iterator> vi;
    int order = 0;
    int indx = 0;
    for(int i = 0; i < index_size; i++)
    {
        s.push_back(scores(i));
    }
    for(vector<float>::iterator it = s.begin(); it != s.end(); it++)
    {
        vi.push_back(it);
    }
    sort(vi.begin(), vi.end(), [](vector<float>::iterator &a,vector<float>::iterator &b) {return *a > *b;});
    for(vector<vector<float>::iterator>::iterator it = vi.begin(); it != vi.end(); it++) {
        indx = *it - s.begin();

        scores_last_p(order) = **it;
        proposals_last_p.chip(order,0) = proposals.chip(indx,0);

        order++;
        if(order >= pre_nms_topN) {
            break;
        }
    }
    Tensor2f proposals_last(order,4);
    Tensor1f scores_last(order);
    Eigen::array<int, 2> off0({0, 0});
    Eigen::array<int, 2> ext0({order, 4});
    proposals_last = proposals_last_p.slice(off0, ext0);
    for(int i = 0; i < order; i++)
        scores_last(i) = scores_last_p(i);
/***********************************************************************/
//NMS的步骤就是对于分数由高到低排序的框，从分数高的开始，看他和后面每一个没有被扔掉的框的IoU是否大于阈值，
//是的话就将后面的这些框扔掉
    Tensor2f p_s(order,5); //形状由pre_nms_topN修改为order
    Eigen::array<int, 2> off1({0, 0});
    Eigen::array<int, 2> ext1({order, 4});  //形状由pre_nms_topN修改为order
    p_s.slice(off1, ext1) = proposals_last;
    p_s.chip(4,1) = scores_last;

    Tensor1i order_nms = gpu_nms(p_s, nms_thresh);
    //std::cout << "order_nms shape = " << order_nms.dimension(0) << std::endl;
    int len = (order_nms.dimension(0) > post_nms_topN) ? post_nms_topN : order_nms.dimension(0);

    Tensor2f proposals_nms(len,4);
    Tensor1f scores_nms(len);

    for (int i = 0; i < len; i++) {
        scores_nms(i) = scores_last(order_nms(i));
        proposals_nms.chip(i,0) = proposals_last.chip(order_nms(i),0);
    }

    Tensor1f batch_inds(len);
    batch_inds.setZero();

    Tensor2f blob(len, 5);
    Eigen::array<int, 2> off2({0, 1});
    Eigen::array<int, 2> ext2({len, 4});
    blob.slice(off2, ext2) = proposals_nms;
    blob.chip(0,1) = batch_inds; //代表图片的index，我们只取一张图片，所以是0

    Tensor *blob_tf = NULL;
    OP_REQUIRES_OK(context, context->allocate_output(0, {len, 5}, &blob_tf));
    cudaMemcpy(blob_tf->flat<float>().data(), blob.data(), len * 5 * sizeof(float), cudaMemcpyHostToHost);
    //std::cout << "proposals_layer output: " << blob_tf->shape().DebugString() << endl;
}

#endif
